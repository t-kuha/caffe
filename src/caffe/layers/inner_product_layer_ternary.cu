#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/inner_product_layer_ternary.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void TernaryInnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // Initialization
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* weight = this->blobs_[0]->mutable_gpu_data();
  const int num = this-> N_;
  const int kel = this-> K_;
  const int N = num * kel;
  Dtype* ternaryweight = ternary_weight_.mutable_gpu_data();
  caffe_copy<Dtype>(N, weight, ternaryweight);
  
  //calculate the mean by kernels
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num, kel,
  1. / N, weight, spatial_sum_multiplier_.gpu_data(), 0.,
        num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemv<Dtype>(CblasTrans, num, 1., 1.,
  num_by_chans_.gpu_data(), batch_sum_multiplier_.gpu_data(), 0.,
        mean_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, 1, 1, 1,
      batch_sum_multiplier_.gpu_data(), mean_.gpu_data(), 0.,
      num_by_chans_.mutable_gpu_data());
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num,
      kel, 1, -0.7 / N, num_by_chans_.gpu_data(),
      spatial_sum_multiplier_.gpu_data(), 0.7 / N, ternaryweight);
  caffe_gpu_asum(N, ternaryweight, delta_.mutable_cpu_data());
  
  // quantize the weights and save the signs into ternaryweight
  caffe_gpu_ternarize<Dtype>(weight, ternaryweight, this->all_quantized_.gpu_data(), delta_.gpu_data(), num, kel);
  caffe_gpu_ternary_scaling<Dtype>(weight, ternaryweight, this->i_delta_weight_.mutable_gpu_data(), this->i_delta_sign_.mutable_gpu_data(),
    this->all_quantized_.gpu_data(), delta_.gpu_data(), &alpha_, num, kel);
  
  // Stochastic Quantization
  if (this->sq_ && (this->ratio_ < 100)){
    // roulette selection algorithm; mask is stored in 'is_quantized'
	Roulette();
    // convert the weights to a hybrid weight
	caffe_gpu_ternarize<Dtype>(weight, ternaryweight, this->is_quantized_.gpu_data(), delta_.gpu_data(), num, kel);
    caffe_gpu_ternary_scaling<Dtype>(weight, ternaryweight, this->i_delta_weight_.mutable_gpu_data(), this->i_delta_sign_.mutable_gpu_data(),
      this->is_quantized_.gpu_data(), delta_.gpu_data(), &alpha_, num, kel);
  }
  
  //Inner Product
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                         ternaryweight, bottom_data, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            this->blobs_[1]->gpu_data(), top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans,
                          transpose_ ? CblasNoTrans : CblasTrans,
                          M_, N_, K_, (Dtype)1.,
                          bottom_data, ternaryweight, (Dtype)0., top_data);
    if (bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), (Dtype)1., top_data);
  }
}

template <typename Dtype>
void TernaryInnerProductLayer<Dtype>::Roulette() {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int num = this-> N_;
  const int weight_col = this-> K_;
  const int N = num * weight_col;
  const Dtype* ternaryweight = ternary_weight_.gpu_data();
  const float ratio = this->ratio_;
  Dtype* norm = error_norm_.mutable_cpu_data();
  Dtype* ns = sum_norm_.mutable_cpu_data();
  Dtype* wc = weight_copy_.mutable_gpu_data();
  
  // calculate the quantization error(||W-Q||/||W||)
  caffe_gpu_sub(N, weight, ternaryweight, wc);
  for(int n = 0; n < num; n++) {
    caffe_gpu_asum(weight_col, wc + n * weight_col, norm + n);
    caffe_gpu_asum(weight_col, weight + n * weight_col, ns + n);
  }
  for(int n = 0; n < num; n++) {
    if (ns[n] == 0) {
      norm[n] = 0;
    } else {
      norm[n] = norm[n] / ns[n]; // quantization errors are stored in 'norm'
    }
  }
  int* is_quant = is_quantized_.mutable_cpu_data();
  
  // roulette
  Dtype sum = 0;
  for(int n = 0; n < num; n++) {
    sum += norm[n];
    is_quant[n] = 1;
  }
  const int real_num = int((1 - ratio / 100) * num);
  for(int i = 0; i < real_num; i++) { // select one kernel which is set to real. the probability is equal to norm
    Dtype p;
    caffe_rng_uniform(1, Dtype(0), Dtype(1), &p);
    p *= sum;
    Dtype cur_sum = 0;
    for(int n = 0; n < num; n++) {
      if(is_quant[n] == 1) { // not selected
        if((p >= cur_sum) && (p < cur_sum + norm[n])) { // hit
          is_quant[n] = 0;
          sum -= norm[n]; // remove
          break;
		}
        else {
          cur_sum += norm[n];
        }
	  }
    }
  }
}

template void TernaryInnerProductLayer<float>::Roulette();
template void TernaryInnerProductLayer<double>::Roulette();

template <typename Dtype>
void TernaryInnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (this->param_propagate_down_[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          K_, N_, M_,
          (Dtype)1., bottom_data, top_diff,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans,
          N_, K_, M_,
          (Dtype)1., top_diff, bottom_data,
          (Dtype)1., this->blobs_[0]->mutable_gpu_diff());
    }
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bias
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
  }
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    // Gradient with respect to bottom data
    if (transpose_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans,
          M_, K_, N_,
          (Dtype)1., top_diff, ternary_weight_.gpu_data(),
          (Dtype)0., bottom[0]->mutable_gpu_diff());
    } else {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans,
          M_, K_, N_,
         (Dtype)1., top_diff, ternary_weight_.gpu_data(),
         (Dtype)0., bottom[0]->mutable_gpu_diff());
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TernaryInnerProductLayer);

}  // namespace caffe
